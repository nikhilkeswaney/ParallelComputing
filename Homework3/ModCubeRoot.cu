
#include <hip/hip_runtime.h>
//******************************************************************************
//
// File:    ModCubeRoot.cu
//
// This CUDA C file is the kernel function for the GPU to try and break the cipher
// key
//
//******************************************************************************

// Number of threads per block.
#define NT 1024
// Overall counter variable in global memory.
__device__ unsigned int incrementNumber;

/**
 * This kernel is used to break the RSA
 * @param  N  is a large integer, e.g. a 2048-bit integer.
 * @param  C  cipher key.
 * @param  possibleValues  array of possible answers.
 *
 * @author  Nikhil Keswaney
 * @version 11-15-2018
 */
extern "C" __global__ void DoBruteForce
   (unsigned long long int N, unsigned long long int C, unsigned long long int* possibleValues)
   {
   unsigned long long int thr, size, rank;
   unsigned long long int m;
   // Determine number of threads and this thread's rank.
   thr = threadIdx.x;
   size = gridDim.x*NT;
   rank = blockIdx.x*NT + thr;

   for (unsigned long long int i = rank; i < N; i += size)
      {
        m = (((i * i) % N) * i) % N;
        if(m == C){
          possibleValues[atomicAdd(&incrementNumber, 1)] = i;        
        }

      }

   }
